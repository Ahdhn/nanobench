#include "hip/hip_runtime.h"
#ifndef _CUGRAPH_
#define _CUGRAPH_


#include "util.cu"
#include <hipblas.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <algorithm>

inline bool moveAndCompare(const int N, double* d_p, double* h_p_gold) {

    std::vector<double> h_p_res(N, 0);
    CUDA_ERROR(hipMemcpy(h_p_res.data(), d_p, N * sizeof(double), hipMemcpyDeviceToHost));
    return compare_arr(h_p_gold, h_p_res.data(), N);
}

inline float DaxpyStream(const int N, double* d_r, double* d_p, const int num_ops,
     double* h_p_gold) {
     //return the time in float 
    double alpha = 1.0;        
    hipStream_t cublas_stream;
    CUDA_ERROR(hipStreamCreate(&cublas_stream));
    hipblasHandle_t cublas_handle = 0;

    CUBLAS_ERROR(hipblasCreate(&cublas_handle));
    CUBLAS_ERROR(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ERROR(hipblasSetStream(cublas_handle, cublas_stream));

    hipEvent_t start, stop;
    CUDA_ERROR(hipEventCreate(&start));
    CUDA_ERROR(hipEventCreate(&stop));
    CUDA_ERROR(hipEventRecord(start, cublas_stream));
    for (int iter = 0; iter < num_ops; ++iter) {
        CUBLAS_ERROR(hipblasDaxpy(cublas_handle, N, &alpha, d_r, 1, d_p, 1));    
        CUDA_ERROR(hipStreamSynchronize(cublas_stream));
    }

    CUDA_ERROR(hipEventRecord(stop, cublas_stream));
    CUDA_ERROR(hipEventSynchronize(stop));
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipGetLastError());    
    CUDA_ERROR(hipStreamDestroy(cublas_stream));
    CUBLAS_ERROR(hipblasDestroy(cublas_handle));

    float time = 0.0f;//ms
    CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
       

    if (!moveAndCompare(N, d_p, h_p_gold)) {
        std::cout << " DaxpyStream() failed with N = " << N << std::endl;
        exit(EXIT_FAILURE);
    }



    return time/ num_ops;

}

inline float DaxpyGraph(const int N, double*d_r, double *d_p, const int num_ops,
                        const int num_nodes, double* h_p_gold) {
    //return the time in float 
    if(num_ops % num_nodes != 0){
        fprintf(stderr, "DaxpyGraph():: num_ops shoudld be divisible by num_nodes");        
        exit(EXIT_FAILURE);
    }
    
    double alpha = 1.0;
    int num_runs = num_ops/num_nodes;
    hipGraph_t cuda_graph;
    hipStream_t capture_stream;
    hipblasHandle_t cublas_handle = 0;
    CUBLAS_ERROR(hipblasCreate(&cublas_handle));
    CUDA_ERROR(hipStreamCreate(&capture_stream));
    CUDA_ERROR(hipStreamBeginCapture(capture_stream, hipStreamCaptureModeGlobal));
    CUBLAS_ERROR(hipblasSetStream(cublas_handle, capture_stream));
    CUBLAS_ERROR(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST));
    for(int n=0;n<num_nodes;++n){
        CUBLAS_ERROR(hipblasDaxpy(cublas_handle, N, &alpha, d_r, 1, d_p, 1));
    }
    CUDA_ERROR(hipStreamEndCapture(capture_stream, &cuda_graph));

    hipGraphNode_t* nodes = NULL;
    size_t generated_num_nodes = 0;
    CUDA_ERROR(hipGraphGetNodes(cuda_graph, nodes, &generated_num_nodes));
    if (generated_num_nodes != num_nodes) {
        fprintf(stderr, "DaxpyGraph():: CUDA Graph has generated %d but the input is %d", 
        static_cast<int>(generated_num_nodes), num_nodes);                
        exit(EXIT_FAILURE);
    }   
    hipGraphExec_t cuda_graph_exec;
    CUDA_ERROR(hipGraphInstantiate(&cuda_graph_exec, cuda_graph, NULL, NULL, 0));

    hipEvent_t start, stop;
    CUDA_ERROR(hipEventCreate(&start));
    CUDA_ERROR(hipEventCreate(&stop));
    CUDA_ERROR(hipEventRecord(start, capture_stream));
    for (int iter = 0; iter < num_runs; ++iter) {
        CUDA_ERROR(hipGraphLaunch(cuda_graph_exec, capture_stream));
        CUDA_ERROR(hipStreamSynchronize(capture_stream));
    }

    CUDA_ERROR(hipEventRecord(stop, capture_stream));
    CUDA_ERROR(hipEventSynchronize(stop));
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipGetLastError());

    CUDA_ERROR(hipGraphExecDestroy(cuda_graph_exec));
    CUDA_ERROR(hipGraphDestroy(cuda_graph));
    CUDA_ERROR(hipStreamDestroy(capture_stream));
    CUBLAS_ERROR(hipblasDestroy(cublas_handle));

    float time = 0.0f;//ms
    CUDA_ERROR(hipEventElapsedTime(&time, start, stop));

    if (!moveAndCompare(N, d_p, h_p_gold)) {
        fprintf(stderr, "DaxpyGraph():: failed with N= %d", N);                
        exit(EXIT_FAILURE);
    }

    return time/ num_ops;
}

inline void benchDriver(const int num_ops, const int num_nodes,
                        const int start, const int end) {

    std::cout << " ****** Bench Driver with "<<  num_nodes<< " nodes Started ******" << std::endl;   
    const char separator = ' ';
    const int numWidth = 15;
    std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "Exp";
    std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "Size";
    std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "GraphTime";
    std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "StreamTime";
    std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "Speedup" << std::endl << std::endl;
    for (int exp = start; exp <= end; ++exp) {
        int N = 1 << exp;        
        double* d_r, * d_p;
        CUDA_ERROR(hipMalloc((void**)&d_r, N * sizeof(double)));
        CUDA_ERROR(hipMalloc((void**)&d_p, N * sizeof(double)));

        std::vector<double> h_r(N, 0);
        std::vector<double> h_p(N, 0);
        
        std::generate(h_r.begin(), h_r.end(), []() {return double(rand()) / double(RAND_MAX); });
        std::generate(h_p.begin(), h_p.end(), []() {return double(rand()) / double(RAND_MAX); });
        
        std::vector<double> h_p_gold(h_p);
        for (int iter = 0; iter < num_ops; ++iter) {
            for (int i = 0; i < N; ++i) {
                h_p_gold[i] = h_r[i] + h_p_gold[i];
            }
        }

        CUDA_ERROR(hipMemcpy(d_r, h_r.data(), N * sizeof(double), hipMemcpyHostToDevice));
        CUDA_ERROR(hipMemcpy(d_p, h_p.data(), N * sizeof(double), hipMemcpyHostToDevice));
        float graph_time = DaxpyGraph(N, d_r, d_p, num_ops, 10, h_p_gold.data());

        CUDA_ERROR(hipMemcpy(d_r, h_r.data(), N * sizeof(double), hipMemcpyHostToDevice));
        CUDA_ERROR(hipMemcpy(d_p, h_p.data(), N * sizeof(double), hipMemcpyHostToDevice));
        float stream_time = DaxpyStream(N, d_r, d_p, num_ops, h_p_gold.data());

        std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << exp;
        std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << N;
        std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << graph_time;
        std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << stream_time;
        std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << stream_time/graph_time << std::endl;        
        

        CUDA_ERROR(hipFree(d_r));
        CUDA_ERROR(hipFree(d_p));
    }
    std::cout << " ****** Bench Driver with "<<  num_nodes<< " nodes Started ******" << std::endl;   
}



#endif // ! _CUGRAPH_