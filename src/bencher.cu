#include "hip/hip_runtime.h"
#ifndef _CUGRAPH_
#define _CUGRAPH_


#include "util.cu"
#include <hipblas.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <algorithm>

inline bool moveAndCompare(const int N, double* d_p, double* h_p_gold) {

    std::vector<double> h_p_res(N, 0);
    CUDA_ERROR(hipMemcpy(h_p_res.data(), d_p, N * sizeof(double), hipMemcpyDeviceToHost));
    return compare_arr(h_p_gold, h_p_res.data(), N);
}


__global__ void static handmadeDaxpy(const int N, double* d_r, double* d_p, double alpha) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;    
    if (tid < N) {
        double p = d_p[tid];
        double r = d_r[tid];
        p *= alpha;
        p += r;
        d_p[tid] = p;
    }
}


/**
* handmadeDaxpyGraph()
*/
inline float handmadeDaxpyGraph(const int N, double* d_r, double* d_p, const int num_ops,
                                const int num_nodes, double* h_p_gold) {
    //return the time in float 
    if (num_ops % num_nodes != 0) {
        fprintf(stderr, "cublasDaxpyGraph():: num_ops should be divisible by num_nodes");
        exit(EXIT_FAILURE);
    }
    int num_runs = num_ops / num_nodes;
    const double alpha = 1.0;
    const int threads = 128;
    const int blocks = (N + threads - 1) / threads;

    hipStream_t stream;
    CUDA_ERROR(hipStreamCreate(&stream));
    
    hipGraph_t graph;
    CUDA_ERROR(hipGraphCreate(&graph, 0));
    hipKernelNodeParams kernel_node_params = { 0 };    
    

    void* kernel_args[4] = { (void*)&N, (void*)&d_r, (void*)&d_p, (void*)&alpha };
    kernel_node_params.func = (void*)handmadeDaxpy;
    kernel_node_params.gridDim = dim3(blocks, 1, 1);
    kernel_node_params.blockDim = dim3(threads, 1, 1);
    kernel_node_params.sharedMemBytes = 0;
    kernel_node_params.kernelParams = (void**)kernel_args;
    kernel_node_params.extra = NULL;    

    std::vector<hipGraphNode_t> kernel_node(num_nodes);
    CUDA_ERROR(hipGraphAddKernelNode(&kernel_node[0], graph, NULL,
        0, &kernel_node_params));
    for (int n = 1; n < num_nodes; ++n) {
        CUDA_ERROR(hipGraphAddKernelNode(&kernel_node[n], graph, &kernel_node[n-1],
            1, &kernel_node_params));
    }

    hipGraphNode_t* nodes = NULL;
    size_t generated_num_nodes = 0;
    CUDA_ERROR(hipGraphGetNodes(graph, nodes, &generated_num_nodes));
    if (generated_num_nodes != num_nodes) {
        fprintf(stderr, "handmadeDaxpyGraph():: CUDA Graph has generated %d but the expected is %d",
            static_cast<int>(generated_num_nodes), num_nodes);
        exit(EXIT_FAILURE);
    }

    hipGraphExec_t exec_graph;
    CUDA_ERROR(hipGraphInstantiate(&exec_graph, graph, NULL, NULL, 0));

    hipEvent_t start, stop;
    CUDA_ERROR(hipEventCreate(&start));
    CUDA_ERROR(hipEventCreate(&stop));
    CUDA_ERROR(hipEventRecord(start, stream));

    for (int iter = 0; iter < num_runs; ++iter) {
        CUDA_ERROR(hipGraphLaunch(exec_graph, stream));        
        CUDA_ERROR(hipStreamSynchronize(stream));
    }

    CUDA_ERROR(hipEventRecord(stop, stream));
    CUDA_ERROR(hipEventSynchronize(stop));
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipGetLastError());
    CUDA_ERROR(hipStreamDestroy(stream));

    float time = 0.0f;//ms
    CUDA_ERROR(hipEventElapsedTime(&time, start, stop));


    if (!moveAndCompare(N, d_p, h_p_gold)) {
        std::cout << " handmadeDaxpyGraph() failed with N = " << N << std::endl;
        exit(EXIT_FAILURE);
    }
    return time / num_ops;
}

/** 
* handmadeDaxpyStream()
*/
inline float handmadeDaxpyStream(const int N, double* d_r, double* d_p, const int num_ops,
    double* h_p_gold) {
    //return the time in float 
    const double alpha = 1.0;
    const int threads = 128;
    const int blocks = (N + threads - 1) / threads;

    hipStream_t stream;
    CUDA_ERROR(hipStreamCreate(&stream));
    
    hipEvent_t start, stop;
    CUDA_ERROR(hipEventCreate(&start));
    CUDA_ERROR(hipEventCreate(&stop));
    CUDA_ERROR(hipEventRecord(start, stream));
    for (int iter = 0; iter < num_ops; ++iter) {
        handmadeDaxpy <<<blocks, threads,0, stream >>>(N, d_r, d_p, alpha);
        CUDA_ERROR(hipStreamSynchronize(stream));
    }

    CUDA_ERROR(hipEventRecord(stop, stream));
    CUDA_ERROR(hipEventSynchronize(stop));
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipGetLastError());
    CUDA_ERROR(hipStreamDestroy(stream));
    
    float time = 0.0f;//ms
    CUDA_ERROR(hipEventElapsedTime(&time, start, stop));


    if (!moveAndCompare(N, d_p, h_p_gold)) {
        std::cout << " handmadeDaxpyStream() failed with N = " << N << std::endl;
        exit(EXIT_FAILURE);
    }
    return time / num_ops;
}

/**
* cublasDaxpyStream()
*/
inline float cublasDaxpyStream(const int N, double* d_r, double* d_p, const int num_ops,
     double* h_p_gold) {
     //return the time in float 
    double alpha = 1.0;        
    hipStream_t cublas_stream;
    CUDA_ERROR(hipStreamCreate(&cublas_stream));
    hipblasHandle_t cublas_handle = 0;

    CUBLAS_ERROR(hipblasCreate(&cublas_handle));
    CUBLAS_ERROR(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ERROR(hipblasSetStream(cublas_handle, cublas_stream));

    hipEvent_t start, stop;
    CUDA_ERROR(hipEventCreate(&start));
    CUDA_ERROR(hipEventCreate(&stop));
    CUDA_ERROR(hipEventRecord(start, cublas_stream));
    for (int iter = 0; iter < num_ops; ++iter) {
        CUBLAS_ERROR(hipblasDaxpy(cublas_handle, N, &alpha, d_r, 1, d_p, 1));    
        CUDA_ERROR(hipStreamSynchronize(cublas_stream));
    }

    CUDA_ERROR(hipEventRecord(stop, cublas_stream));
    CUDA_ERROR(hipEventSynchronize(stop));
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipGetLastError());    
    CUDA_ERROR(hipStreamDestroy(cublas_stream));
    CUBLAS_ERROR(hipblasDestroy(cublas_handle));

    float time = 0.0f;//ms
    CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
       

    if (!moveAndCompare(N, d_p, h_p_gold)) {
        std::cout << " cublasDaxpyStream() failed with N = " << N << std::endl;
        exit(EXIT_FAILURE);
    }



    return time/ num_ops;

}

/**
* cublasDaxpyGraph()
*/
inline float cublasDaxpyGraph(const int N, double*d_r, double *d_p, const int num_ops,
                        const int num_nodes, double* h_p_gold) {
    //return the time in float 
    if(num_ops % num_nodes != 0){
        fprintf(stderr, "cublasDaxpyGraph():: num_ops should be divisible by num_nodes");        
        exit(EXIT_FAILURE);
    }
    
    double alpha = 1.0;
    int num_runs = num_ops/num_nodes;
    hipGraph_t cuda_graph;
    hipStream_t capture_stream;
    hipblasHandle_t cublas_handle = 0;
    CUBLAS_ERROR(hipblasCreate(&cublas_handle));
    CUDA_ERROR(hipStreamCreate(&capture_stream));
    CUDA_ERROR(hipStreamBeginCapture(capture_stream, hipStreamCaptureModeGlobal));
    CUBLAS_ERROR(hipblasSetStream(cublas_handle, capture_stream));
    CUBLAS_ERROR(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST));
    for(int n=0;n<num_nodes;++n){
        CUBLAS_ERROR(hipblasDaxpy(cublas_handle, N, &alpha, d_r, 1, d_p, 1));
    }
    CUDA_ERROR(hipStreamEndCapture(capture_stream, &cuda_graph));

    hipGraphNode_t* nodes = NULL;
    size_t generated_num_nodes = 0;
    CUDA_ERROR(hipGraphGetNodes(cuda_graph, nodes, &generated_num_nodes));
    if (generated_num_nodes != num_nodes) {
        fprintf(stderr, "cublasDaxpyGraph():: CUDA Graph has generated %d but the expected is %d",        
        static_cast<int>(generated_num_nodes), num_nodes);                
        exit(EXIT_FAILURE);
    }   
    hipGraphExec_t cuda_graph_exec;
    CUDA_ERROR(hipGraphInstantiate(&cuda_graph_exec, cuda_graph, NULL, NULL, 0));

    hipEvent_t start, stop;
    CUDA_ERROR(hipEventCreate(&start));
    CUDA_ERROR(hipEventCreate(&stop));
    CUDA_ERROR(hipEventRecord(start, capture_stream));
    for (int iter = 0; iter < num_runs; ++iter) {
        CUDA_ERROR(hipGraphLaunch(cuda_graph_exec, capture_stream));
        CUDA_ERROR(hipStreamSynchronize(capture_stream));
    }

    CUDA_ERROR(hipEventRecord(stop, capture_stream));
    CUDA_ERROR(hipEventSynchronize(stop));
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipGetLastError());

    CUDA_ERROR(hipGraphExecDestroy(cuda_graph_exec));
    CUDA_ERROR(hipGraphDestroy(cuda_graph));
    CUDA_ERROR(hipStreamDestroy(capture_stream));
    CUBLAS_ERROR(hipblasDestroy(cublas_handle));

    float time = 0.0f;//ms
    CUDA_ERROR(hipEventElapsedTime(&time, start, stop));

    if (!moveAndCompare(N, d_p, h_p_gold)) {
        fprintf(stderr, "cublasDaxpyGraph():: failed with N= %d", N);                
        exit(EXIT_FAILURE);
    }

    return time/ num_ops;
}

/**
* benchDriver()
*/
inline void benchDriver(const int num_ops, const int num_nodes,
                        const int start, const int end) {

    std::cout << " ****** Bench Driver with " << num_ops << " operations and "
        << num_nodes << " nodes Started ******" << std::endl;
    const char separator = ' ';
    const int numWidth = 20;
    std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "Exp (2^x)";
    std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "Size";
    std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "CUBLAS GraphTime";
    std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "CUBLAS StreamTime";
    std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "HANDMADE GraphTime";
    std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "HANDMADE StreamTime";
    //std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << "Speedup";
    std::cout << std::endl << std::endl;

    for (int exp = start; exp <= end; ++exp) {
        int N = 1 << exp;                
        double* d_r, * d_p;
        CUDA_ERROR(hipMalloc((void**)&d_r, N * sizeof(double)));
        CUDA_ERROR(hipMalloc((void**)&d_p, N * sizeof(double)));

        std::vector<double> h_r(N, 0);
        std::vector<double> h_p(N, 0);
        
        std::generate(h_r.begin(), h_r.end(), []() {return double(rand()) / double(RAND_MAX); });
        std::generate(h_p.begin(), h_p.end(), []() {return double(rand()) / double(RAND_MAX); });
        
        std::vector<double> h_p_gold(h_p);
        for (int iter = 0; iter < num_ops; ++iter) {
            for (int i = 0; i < N; ++i) {
                h_p_gold[i] = h_r[i] + h_p_gold[i];
            }
        }

        CUDA_ERROR(hipMemcpy(d_r, h_r.data(), N * sizeof(double), hipMemcpyHostToDevice));
        CUDA_ERROR(hipMemcpy(d_p, h_p.data(), N * sizeof(double), hipMemcpyHostToDevice));
        float cublas_graph_time = cublasDaxpyGraph(N, d_r, d_p, num_ops, num_nodes, h_p_gold.data());

        CUDA_ERROR(hipMemcpy(d_r, h_r.data(), N * sizeof(double), hipMemcpyHostToDevice));
        CUDA_ERROR(hipMemcpy(d_p, h_p.data(), N * sizeof(double), hipMemcpyHostToDevice));
        float cublas_stream_time = cublasDaxpyStream(N, d_r, d_p, num_ops, h_p_gold.data());
        
        CUDA_ERROR(hipMemcpy(d_r, h_r.data(), N * sizeof(double), hipMemcpyHostToDevice));
        CUDA_ERROR(hipMemcpy(d_p, h_p.data(), N * sizeof(double), hipMemcpyHostToDevice));
        float handmade_graph_time = handmadeDaxpyGraph(N, d_r, d_p, num_ops, num_nodes, h_p_gold.data());

        CUDA_ERROR(hipMemcpy(d_r, h_r.data(), N * sizeof(double), hipMemcpyHostToDevice));
        CUDA_ERROR(hipMemcpy(d_p, h_p.data(), N * sizeof(double), hipMemcpyHostToDevice));
        float handmade_stream_time = handmadeDaxpyStream(N, d_r, d_p, num_ops, h_p_gold.data());

        std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << exp;
        std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << N;
        std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << cublas_graph_time;
        std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << cublas_stream_time;
        std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << handmade_graph_time;
        std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << handmade_stream_time;
        //std::cout << std::left << std::setw(numWidth) << std::setfill(separator) << cublas_stream_time/ cublas_graph_time;
        std::wcout << std::endl;
        

        CUDA_ERROR(hipFree(d_r));
        CUDA_ERROR(hipFree(d_p));
    }
    std::cout << " ****** Bench Driver with " << num_ops << " operations and "
        << num_nodes << " nodes Ended ******" << std::endl;    
}



#endif // ! _CUGRAPH_